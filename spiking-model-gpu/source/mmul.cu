#include "hip/hip_runtime.h"
// This program computes matrix multiplication using shared memory tiling
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

#include "mmul.h"

__global__ void matrixMul(const int *a, const int *b, int *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Statically allocated shared memory
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Accumulate in temporary variable
  int tmp = 0;

  // Sweep tile across matrix
  for (int i = 0; i < N; i += blockDim.x) {
    // Load in elements for this tile
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];

    // Wait for both tiles to be loaded in before doing computation
    __syncthreads();

    // Do matrix multiplication on the small matrix
    for (int j = 0; j < blockDim.x; j++) {
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }

    // Wait for all threads to finish using current tiles before loading in new
    // ones
    __syncthreads();
  }

  // Write back results
  c[row * N + col] = tmp;
}

void matrixMultiply(const int *a, const int *b, int *c)
{
  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  matrixMul<<<blocks, threads>>>(a, b, c);
}
